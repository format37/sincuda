#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "../../common/book.h"

__global__ void kernel(float *resless, float *res)
{
	//float res[90*314];
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	float real, patt, x;
	x = (float)tid / 100;
	real = (1.2*sin(5.01*x) + 2.3*sin(17 * x)) / (1.2 + 2.3);
	//calculate
	patt = sin((float)bid*x);
	res[bid*314+tid] = (real > patt ? real - patt : patt - real);
	__syncthreads();
	//reduce to self
	int index = 314 / 2;
	while (index != 0)
	{
		if (tid < index) res[bid * 314 + tid] += res[bid * 314 + tid + index];
		__syncthreads();
		index /= 2;
	}
	//reduce less
	if (tid == 0&&bid<90) resless[bid] = res[bid*314];
}

int main(void)
{
	float resless[90],*dev_resless,res[90*314],*dev_res;
	//float sig[314];
	/*hipEvent_t     start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));*/
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffer
	cudaStatus = hipMalloc((void**)&dev_resless, 90 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_res, 314*90 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//// Copy input vectors from host memory to GPU buffers.
	//cudaStatus = hipMemcpy(dev_res, res, 90 * sizeof(float), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//	goto Error;
	//}
	//sinnatvol++
	kernel << <90,314 >> >(dev_resless,dev_res);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(resless, dev_resless, 90 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(res, dev_res, 314*90 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//for (x = 0; x < 3.14; x = x + 0.01)
	/*for (xj = 0; xj < 314; xj ++)
	{
		x = xj / 100;
		real = (1.2*sin(5.01*x) + 2.3*sin(17 * x)) / (1.2 + 2.3);
		for (i = 1; i < 90; i++)
		{
			patt = sin(i*x);
			res[i] = res[i] + (real > patt ? real - patt : patt - real);
		}
	}*/

	//sinnatvol--
	// get stop time, and display the timing results
	/*HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float   elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
		start, stop));
	printf("Time to generate:  %3.1f ms\n", elapsedTime);

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));*/

	// free memory on the gpu side
	cudaStatus = hipFree(dev_resless);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipFree failed!");
		goto Error;
	}
	cudaStatus = hipFree(dev_res);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipFree failed!");
		goto Error;
	}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	// free memory on the cpu side
	free(resless);
	//free(res);

    return 0;

Error:
	hipFree(dev_resless);

	return cudaStatus;
}
